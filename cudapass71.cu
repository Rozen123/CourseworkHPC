#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
/****************************************************************************
  This program gives an example of a poor way to implement a password cracker
  in CUDA C. It is poor because it acheives this with just one thread, which
  is obviously not good given the scale of parallelism available to CUDA
  programs.
  
  The intentions of this program are:
    1) Demonstrate the use of __device__ and __global__ functions
    2) Enable a simulation of password cracking in the absence of library 
       with equivalent functionality to libcrypt. The password to be found
       is hardcoded into a function called is_a_match.   

  Compile and run with:
    nvcc -o cudapass71 cudapass71.cu
    ./cudapass71
   
  Dr Kevan Buckley, University of Wolverhampton, 2018
*****************************************************************************/

/****************************************************************************
  This function returns 1 if the attempt at cracking the password is 
  identical to the plain text password string stored in the program. 
  Otherwise,it returns 0.
*****************************************************************************/

__device__ int is_a_match(char *attempt) {
	char plain_password1[] = "AB1234";
	char plain_password2[] = "CD2345";
	char plain_password3[] = "EF3456";
	char plain_password4[] = "GH4567";


	char *b = attempt;
	char *i = attempt;
	char *n = attempt;
	char *o = attempt;
	char *b1 = plain_password1;
	char *i2 = plain_password2;
	char *n3 = plain_password3;
	char *o4 = plain_password4;

	while(*b == *b1) { 
		if(*b == '\0') 
		{
			printf("Password: %s\n",plain_password1);
			break;
		}

		b++;
		b1++;
	}
	
	while(*i == *i2) { 
		if(*i == '\0') 
		{
			printf("Password: %s\n",plain_password2);
			break;
		}

		i++;
		i2++;
	}

	while(*n == *n3) { 
		if(*n == '\0') 
		{
			printf("Password: %s\n",plain_password3);
			break;
		}

		n++;
		n3++;
	}

	while(*o == *o4) { 
		if(*o == '\0') 
		{
			printf("Password: %s\n",plain_password4);
			return 1;
		}

		o++;
		o4++;
	}
	return 0;

}

__global__ void  kernel() {
	char i1,i2,i3,i4;

	char password[7];
	password[6] = '\0';

	int i = blockIdx.x+65;
	int j = threadIdx.x+65;
	char firstMatch = i; 
	char secondMatch = j; 

	password[0] = firstMatch;
	password[1] = secondMatch;
	for(i1='0'; i1<='9'; i1++){
		for(i2='0'; i2<='9'; i2++){
			for(i3='0'; i3<='9'; i3++){
				for(i4='0'; i4<='9'; i4++){
					password[2] = i1;
					password[3] = i2;
					password[4] = i3;
					password[5] = i4; 
					if(is_a_match(password)) {
					} 
					else {
	     			//printf("tried: %s\n", password);		  
					}
				}
			}
		}
	}
}

// Calculate the difference between two times. Returns zero on
// success and the time difference through an argument. It will 
// be unsuccessful if the start time is after the end time.

int time_difference(struct timespec *start, 
                    struct timespec *finish, 
                    long long int *difference) {
  long long int d_sec =  finish->tv_sec - start->tv_sec; 
  long long int d_nsec =  finish->tv_nsec - start->tv_nsec; 

  if(d_nsec < 0 ) {
    d_sec--;
    d_nsec += 1000000000; 
  } 
  *difference = d_sec * 1000000000 + d_nsec;
  return !(*difference > 0);
}


int main() {

	struct  timespec start, finish;
	long long int time_elapsed;
	clock_gettime(CLOCK_MONOTONIC, &start);

	kernel <<<26,26>>>();
	hipDeviceSynchronize();

	clock_gettime(CLOCK_MONOTONIC, &finish);
	time_difference(&start, &finish, &time_elapsed);
	printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9)); 

	return 0;
}


